#include <hip/hip_runtime.h>

// https://forums.developer.nvidia.com/t/how-to-understand-the-following-asm/288932
__noinline__ __device__ short2 foo(int a, int b)
{
    short2 res;
    res.x  = a;
    res.y  = b;
    return res;
}