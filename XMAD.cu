#include <hip/hip_runtime.h>

__noinline__ __device__ unsigned foo()
{
    unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    return tid;
}